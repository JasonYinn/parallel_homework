#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <time.h>
#include <math.h>
#include "svjpeg.hpp"

struct render_params
{
    float cx;
    float cy;
    float fx;
    float fy;
    int H;
    int W;
    int dimx, dimy, dimz;
    float origins[3];
    float voxel_size;
};

template <typename scalar_t>
int readBin(const char* fname, scalar_t *arr, int len) {
    std::ifstream in(fname, std::ios::in | std::ios::binary);

    in.read((char *) arr, sizeof(scalar_t) * len);

    return in.gcount();
}

__device__ __forceinline__ float dot(float3 vec1, float3 vec2) {
    return vec1.x * vec2.x + vec1.y * vec2.y + vec1.z * vec2.z;
}

__device__ __forceinline__ float norm(float3 vec) {
    return sqrtf32(vec.x * vec.x + vec.y * vec.y + vec.z * vec.z);
}

__global__
void generate_ray_per_pixel(float *rays, float *c2w, render_params renderParams) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= 0 && i < renderParams.H && j >= 0 && j < renderParams.W){    
        float3 dirs = make_float3(((float)j - renderParams.cx) / renderParams.fx, -((float)i - renderParams.cy) / renderParams.fy, -1.0);
        float3 pose_line;
        for (int k = 0; k < 3; ++k) {
            rays[i * renderParams.W * 8 + j * 8 + k] = c2w[k * 4 + 3];
            pose_line = make_float3(c2w[k * 4 + 0], c2w[k * 4 + 1], c2w[k * 4 + 2]);
            // for (int z = 0; z < 3; ++z) {
            //     temp += dirs[z] * c2w[k * 4 + z];
            // }
            rays[i * renderParams.W * 8 + j * 8 + 4 + k] = dot(dirs, pose_line);
        }

        float norm = sqrt(rays[i * renderParams.W * 8 + j * 8 + 4 + 0] * rays[i * renderParams.W * 8 + j * 8 + 4 + 0] + 
                        rays[i * renderParams.W * 8 + j * 8 + 4 + 1] * rays[i * renderParams.W * 8 + j * 8 + 4 + 1] + 
                        rays[i * renderParams.W * 8 + j * 8 + 4 + 2] * rays[i * renderParams.W * 8 + j * 8 + 4 + 2]);
        rays[i * renderParams.W * 8 + j * 8 + 4 + 0] = rays[i * renderParams.W * 8 + j * 8 + 4 + 0] / norm;
        rays[i * renderParams.W * 8 + j * 8 + 4 + 1] = rays[i * renderParams.W * 8 + j * 8 + 4 + 1] / norm;
        rays[i * renderParams.W * 8 + j * 8 + 4 + 2] = rays[i * renderParams.W * 8 + j * 8 + 4 + 2] / norm;
        __syncthreads();
    }
}

__device__
float3 mul_scalar(float3 vec, float scalar) {
    return make_float3(vec.x * scalar, vec.y * scalar, vec.z * scalar);
}

__device__ __forceinline__
float3 add_vec(float3 vec1, float3 vec2) {
    return make_float3(vec1.x + vec2.x, vec1.y + vec2.y, vec1.z + vec2.z);
}

__device__ __forceinline__
int3 add_int3(int3 vec1, int3 vec2) {
    return make_int3(vec1.x + vec2.x, vec1.y + vec2.y, vec1.z + vec2.z);
}

__device__ __forceinline__ 
int3 pts2coords(float3 pts, float3 origins, float voxel_size) {
    return make_int3(int(floor(pts.x - origins.x) / voxel_size), int(floor(pts.y - origins.y) / voxel_size), int(floor(pts.z - origins.z) / voxel_size));
}

__device__ __forceinline__ 
float3 pts2weights(float3 pts, float3 origins, int3 coords, float voxel_size) {
    return make_float3((pts.x - origins.x) / voxel_size - float(coords.x), (pts.y - origins.y) / voxel_size - float(coords.y), (pts.z - origins.z) / voxel_size - float(coords.z));
}

__global__
void render_rays(const float *rays, const float *volume, uint8_t *imgs, render_params renderParams, float iso_val, float threshold, float max_depth, float stride, int samples) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (i >= 0 && i < renderParams.H && j >= 0 && j < renderParams.W) {
        float3 ray_o = make_float3(rays[i * renderParams.W * 8 + j * 8 + 0], rays[i * renderParams.W * 8 + j * 8 + 1], rays[i * renderParams.W * 8 + j * 8 + 2]);
        float3 ray_d = make_float3(rays[i * renderParams.W * 8 + j * 8 + 4], rays[i * renderParams.W * 8 + j * 8 + 5], rays[i * renderParams.W * 8 + j * 8 + 6]);
        float3 origin = make_float3(renderParams.origins[0], renderParams.origins[1], renderParams.origins[2]);

        const int3 offsets[8] = {make_int3(0, 0, 0), make_int3(0, 0, 1), make_int3(0, 1, 0), make_int3(0, 1, 1),
                                 make_int3(1, 0, 0), make_int3(1, 0, 1), make_int3(1, 1, 0), make_int3(1, 1, 1)};
        // const int offsets[8][3] = {{0, 0, 0}, {0, 0, 1}, {0, 1, 0}, {0, 1, 1},
        //                           {1, 0, 0}, {1, 0, 1}, {1, 1, 0}, {1, 1, 1}};

        float3 pts, weights;
        int3 coords, tmp_coords;
        float ray_len = 0., tsdf = 0.;
        float3 color = make_float3(0., 0., 0.), tmp_color;
        float colors[3] = {0., 0., 0.};
        int oneDim_coord;
        float color_weight;
        
        for (int k = 1; k < samples; ++k) {
            tsdf = 0.;
            ray_len += stride;
            pts.x = ray_o.x + ray_len * ray_d.x;
            pts.y = ray_o.y + ray_len * ray_d.y;
            pts.z = ray_o.z + ray_len * ray_d.z;
            // ray_d = mul_scalar(ray_d, ray_len);
            // pts = add_vec(ray_o, ray_d);

            coords.x = int(floor((pts.x - origin.x) / renderParams.voxel_size));
            coords.y = int(floor((pts.y - origin.y) / renderParams.voxel_size));
            coords.z = int(floor((pts.z - origin.z) / renderParams.voxel_size));
            // coords = pts2coords(pts, origin, renderParams.voxel_size);
            // printf ("[%d, %d]: (%f, %f, %f), (%d, %d, %d)\n", i, j, pts.x, pts.y, pts.x, coords.x, coords.y, coords.z);
            
            weights.x = (pts.x - origin.x) / renderParams.voxel_size - float(coords.x);
            weights.y = (pts.y - origin.y) / renderParams.voxel_size - float(coords.y);
            weights.z = (pts.z - origin.z) / renderParams.voxel_size - float(coords.z);
            // weights = pts2weights(pts, origin, coords, renderParams.voxel_size);
            // printf ("[%d, %d]: (%f, %f, %f), (%f, %f, %f)\n", i, j, pts.x, pts.y, pts.x, weights.x, weights.y, weights.z);

            for (int offset = 0; offset < 8; ++offset) {
                tmp_coords.x = coords.x + offsets[offset].x;
                tmp_coords.y = coords.y + offsets[offset].y;
                tmp_coords.z = coords.z + offsets[offset].z;
                // tmp_coords = add_int3(coords, offsets[offset]);
                if (tmp_coords.x >= 0 && tmp_coords.x < renderParams.dimx && 
                    tmp_coords.y >= 0 && tmp_coords.y < renderParams.dimy &&
                    tmp_coords.z >= 0 && tmp_coords.z < renderParams.dimz) {
                        tsdf += abs(float(offsets[offset].x) - weights.x) * 
                                abs(float(offsets[offset].y) - weights.y) *
                                abs(float(offsets[offset].z) - weights.z) * 
                                volume[tmp_coords.x * renderParams.dimy * renderParams.dimz * 4 + tmp_coords.y * renderParams.dimz * 4 + tmp_coords.z * 4];
                    }
            }
            // printf ("[%d, %d]: %f, %f, %f, %f\n", i, j, pts.x, pts.y, pts.x, tsdf);
            if (abs(tsdf - iso_val) <= threshold) {
                color = make_float3(0., 0., 0.);
                for (int offset = 0; offset < 8; ++offset) {
                tmp_coords.x = coords.x + offsets[offset].x;
                tmp_coords.y = coords.y + offsets[offset].y;
                tmp_coords.z = coords.z + offsets[offset].z;
                // tmp_coords = add_int3(coords, offsets[offset]);
                if (tmp_coords.x >= 0 && tmp_coords.x < renderParams.dimx && 
                    tmp_coords.y >= 0 && tmp_coords.y < renderParams.dimy &&
                    tmp_coords.z >= 0 && tmp_coords.z < renderParams.dimz) {
                        oneDim_coord = tmp_coords.x * renderParams.dimy * renderParams.dimz * 4 + tmp_coords.y * renderParams.dimz * 4 + tmp_coords.z * 4;
                        
                        color_weight = abs(float(offsets[offset].x) - weights.x) * 
                                       abs(float(offsets[offset].y) - weights.y) *
                                       abs(float(offsets[offset].z) - weights.z);
                        // printf ("[%d, %d]: (%d, %d, %d), (%d, %d, %d), %f\n", i, j, tmp_coords.x, tmp_coords.y, tmp_coords.z, coords.x, coords.y, coords.z, color_weight);
                        tmp_color = make_float3(volume[oneDim_coord + 1], 
                                                volume[oneDim_coord + 2],
                                                volume[oneDim_coord + 3]);
                        // printf ("[%d, %d]: %f, %f, %f, %f\n", i, j, tmp_color.x, tmp_color.y, tmp_color.x, color_weight);
                        // color.x = color.x + volume[oneDim_coord + 1] * color_weight;
                        // color.y = color.y + volume[oneDim_coord + 2] * color_weight;
                        // color.z = color.z + volume[oneDim_coord + 3] * color_weight;
                        tmp_color = mul_scalar(tmp_color, color_weight);
                        color = add_vec(color, tmp_color);
                    }
                }
                // color.x = volume[coords.x * renderParams.dimy * renderParams.dimz * 4 + coords.y * renderParams.dimz * 4 + coords.z * 4 + 1];
                // color.y = volume[coords.x * renderParams.dimy * renderParams.dimz * 4 + coords.y * renderParams.dimz * 4 + coords.z * 4 + 2];
                // color.z = volume[coords.x * renderParams.dimy * renderParams.dimz * 4 + coords.y * renderParams.dimz * 4 + coords.z * 4 + 3];
                // printf ("[%d, %d]: (%f, %f, %f)\n", i, j, color.z, color.y, color.x);
                imgs[i * renderParams.W * 3 + j * 3 + 0] = uint8_t(color.z);
                imgs[i * renderParams.W * 3 + j * 3 + 1] = uint8_t(color.y);
                imgs[i * renderParams.W * 3 + j * 3 + 2] = uint8_t(color.x);
                break;
                // printf ("[%d, %d]: %f, %f, %f\n", i, j, color.z, color.y, color.x);
            }
        }
        // imgs[i * renderParams.W * 3 + j * 3 + 0] = int(color.z);
        // imgs[i * renderParams.W * 3 + j * 3 + 1] = int(color.y);
        // imgs[i * renderParams.W * 3 + j * 3 + 2] = int(color.x);
        __syncthreads();
    }
}

void scale_intr(float *intr, int origin_w, int origin_h, int target_w, int target_h) {
    float scale_w = float(origin_w) / float(target_w);
    float scale_h = float(origin_h) / float(target_h);

    for (int i = 0; i < 4; ++i) {
        intr[i] = intr[i] / scale_w;
        intr[i + 4] = intr[i + 4] / scale_h;
    }
}

int main(int argc, char* argv[]) {
    float frac = 1.0;
    int BLOCKSIZE = 8;

    if (argc > 1) {
        frac = atof(argv[1]);
    }
    if (argc > 2) {
        BLOCKSIZE = atoi(argv[2]);
    }

    const char pose_path[] = "./data_bin/pose.bin";
    const char intr_path[] = "./data_bin/intrinsic_color.bin";
    const char info_path[] = "./data_bin/render_info.bin";
    const char vol_path[]  = "./data_bin/volume.bin";
    const int origin_w = 1296;
    const int origin_h = 968;
    const int target_w = (int)(frac * origin_w);
    const int target_h = (int)(frac * origin_h);
    const int samples = 80;
    const float max_depth = 3.;
    const float stride = max_depth / float(samples);
    const float iso_val = 0.;
    const float threshold = 0.5;

    float *info;
    hipMallocManaged((void **)&info, 9 * sizeof(float));
    int bytes = readBin(info_path, info, 9);
    // std::cout << bytes << " bytes of render info have been read" << std::endl;

    const int dims[3] = {(int)info[4], (int)info[5], (int)info[6]};
    const float origins[3] = {info[0], info[1], info[2]};
    const float voxel_size = info[3];
    const int img_size[2] = {(int)info[7], (int)info[8]};

    float *pose;
    hipMallocManaged((void **)&pose, 16 * sizeof(float));
    bytes = readBin(pose_path, pose, 16);
    // std::cout << bytes << " bytes of pose have been read" << std::endl;

    float *intr;
    hipMallocManaged((void **)&intr, 16 * sizeof(float));
    bytes = readBin(intr_path, intr, 16);
    // std::cout << bytes << " bytes of intrinsic have been read" << std::endl;
    scale_intr(intr, origin_w, origin_h, target_w, target_h);

    float *vol;
    hipMallocManaged((void **)&vol, dims[0] * dims[1] * dims[2] * 4 * sizeof(float));
    bytes = readBin(vol_path, vol, dims[0] * dims[1] * dims[2] * 4);
    // std::cout << bytes << " bytes of volume have been read" << std::endl;
    
    float *rays;
    hipMallocManaged((void **)&rays, target_h * target_w * 8 * sizeof(float));

    uint8_t *img, *img_cpu = new uint8_t[target_h * target_w * 3];
    hipMallocManaged((void **)&img, target_h * target_w * 3 * sizeof(uint8_t));

    int device = 0;
    hipMemPrefetchAsync((void *)info, 9 * sizeof(float), device, NULL);
    hipMemPrefetchAsync((void *)pose, 16 * sizeof(float), device, NULL);
    hipMemPrefetchAsync((void *)intr, 16 * sizeof(float), device, NULL);
    hipMemPrefetchAsync((void *)vol, dims[0] * dims[1] * dims[2] * 4 * sizeof(float), device, NULL);
    hipMemPrefetchAsync((void *)rays, target_h * target_w * 8 * sizeof(float), device, NULL);
    hipMemPrefetchAsync((void *)img, target_h * target_w * 3 * sizeof(uint8_t), device, NULL);

    render_params renderParams;
    renderParams.cx = intr[2];
    renderParams.cy = intr[6];
    renderParams.fx = intr[0];
    renderParams.fy = intr[5];
    renderParams.H = target_h;
    renderParams.W = target_w;
    renderParams.dimx = dims[0];
    renderParams.dimy = dims[1];
    renderParams.dimz = dims[2];
    renderParams.origins[0] = origins[0];
    renderParams.origins[1] = origins[1];
    renderParams.origins[2] = origins[2];
    renderParams.voxel_size = voxel_size;

    const dim3 gridShape((target_h + BLOCKSIZE - 1) / BLOCKSIZE, (target_w + BLOCKSIZE - 1) / BLOCKSIZE);
    const dim3 blockShape(BLOCKSIZE, BLOCKSIZE);

    struct timespec sts, ets;
    timespec_get(&sts, TIME_UTC);

    generate_ray_per_pixel<<<gridShape, blockShape>>>(rays, pose, renderParams);
    hipDeviceSynchronize();

    timespec_get(&ets, TIME_UTC);
    time_t dsec = ets.tv_sec - sts.tv_sec;
    long dnsec = ets.tv_nsec - sts.tv_nsec;
    printf ("image size [%d, %d], block size %d, generate rays: %ld.%09lds\n", target_h, target_w, BLOCKSIZE, dsec, dnsec);

    timespec_get(&sts, TIME_UTC);
    
    render_rays<<<gridShape, blockShape>>>(rays, vol, img, renderParams, iso_val, threshold, max_depth, stride, samples);
    hipDeviceSynchronize();

    timespec_get(&ets, TIME_UTC);
    dsec = ets.tv_sec - sts.tv_sec;
    dnsec = ets.tv_nsec - sts.tv_nsec;
    printf ("image size: [%d, %d], block size %d, render rays: %ld.%09lds\n", target_h, target_w, BLOCKSIZE, dsec, dnsec);

    FILE *fp = fopen("./test_cuda.jpeg", "wb");
    svjpeg(fp, target_w, target_h, img);

    hipFree(info);
    hipFree(pose);
    hipFree(intr);
    hipFree(vol);
    hipFree(rays);
    hipFree(img);
    return 0;
}